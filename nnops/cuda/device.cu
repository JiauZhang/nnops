#include <nnops/device.h>
#include <nnops/cuda/device.h>
#include <hip/hip_runtime.h>

namespace nnops::cuda {

int Device::device_count_ = -1;
int Device::warp_size_ = -1;
int Device::max_threads_per_block_ = -1;
int Device::max_threads_dim_[3];
int Device::max_grid_size_[3];

Device::Device() {
    if (device_count_ >= 0)
        return;
    hipGetDeviceCount(&device_count_);
    if (device_count_ > 0) {
        hipDeviceProp_t dp;
        hipGetDeviceProperties(&dp, 0);
        warp_size_ = dp.warpSize;
        max_threads_per_block_ = dp.maxThreadsPerBlock;
        for (int i=0; i<3; i++) {
            max_threads_dim_[i] = dp.maxThreadsDim[i];
            max_grid_size_[i] = dp.maxGridSize[i];
        }
    } else {
        device_count_ = 0;
    }
}

void *Device::malloc(size_t size) {
    void *cuda_mem_ptr = nullptr;
    hipMalloc(&cuda_mem_ptr, size);
    return cuda_mem_ptr;
}

void Device::free(void *ptr) {
    hipFree(ptr);
}

void Device::copy_to_cpu(void *src, void *dst, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

void Device::copy_from_cpu(void *src, void *dst, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

} // namespace nnops::cuda