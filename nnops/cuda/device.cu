#include "hip/hip_runtime.h"
#include <nnops/device.h>
#include <nnops/cuda/device.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

namespace nnops::cuda {

int Device::device_count_ = -1;
int Device::multiprocessor_count_ = -1;
int Device::max_threads_per_multiprocessor_ = -1;
int Device::warp_size_ = -1;
int Device::max_threads_per_block_ = -1;
int Device::max_threads_dim_[3];
int Device::max_grid_size_[3];

Device::Device() {
    if (device_count_ >= 0)
        return;
    hipGetDeviceCount(&device_count_);
    if (device_count_ > 0) {
        hipDeviceProp_t dp;
        hipGetDeviceProperties(&dp, 0);
        multiprocessor_count_ = dp.multiProcessorCount;
        max_threads_per_multiprocessor_ = dp.maxThreadsPerMultiProcessor;
        warp_size_ = dp.warpSize;
        max_threads_per_block_ = dp.maxThreadsPerBlock;
        for (int i=0; i<3; i++) {
            max_threads_dim_[i] = dp.maxThreadsDim[i];
            max_grid_size_[i] = dp.maxGridSize[i];
        }
    } else {
        device_count_ = 0;
    }
}

void Device::info() {
    printf("device count: %d\n", device_count_);
    printf("multiprocessor count: %d\n", multiprocessor_count_);
    printf("max threads per multiprocessor: %d\n", max_threads_per_multiprocessor_);
    printf("warp size: %d\n", warp_size_);
    printf("max threads per block: %d\n", max_threads_per_block_);
    auto *_x = max_threads_dim_;
    printf("max threads dim: %d, %d, %d\n", _x[0], _x[1], _x[2]);
    _x = max_grid_size_;
    printf("max grid size: %d, %d, %d\n", _x[0], _x[1], _x[2]);
}

void *Device::malloc(size_t size) {
    void *cuda_mem_ptr = nullptr;
    hipMalloc(&cuda_mem_ptr, size);
    return cuda_mem_ptr;
}

void Device::free(void *ptr) {
    hipFree(ptr);
}

void Device::copy_to_cpu(void *src, void *dst, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

void Device::copy_from_cpu(void *src, void *dst, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

} // namespace nnops::cuda