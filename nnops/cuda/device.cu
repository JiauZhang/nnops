#include <nnops/device.h>
#include <nnops/cuda/device.h>
#include <hip/hip_runtime.h>

namespace nnops::cuda {

int Device::device_count_ = -1;

Device::Device() {
    if (device_count_ >= 0)
        return;
    hipGetDeviceCount(&device_count_);
}

void *Device::malloc(size_t size) {
    void *cuda_mem_ptr = nullptr;
    hipMalloc(&cuda_mem_ptr, size);
    return cuda_mem_ptr;
}

void Device::free(void *ptr) {
    hipFree(ptr);
}

void Device::copy_to_cpu(void *src, void *dst, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

void Device::copy_from_cpu(void *src, void *dst, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

} // namespace nnops::cuda